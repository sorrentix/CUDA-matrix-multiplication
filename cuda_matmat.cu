#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
extern "C" {
	#include "c_timer.h"
}

int main(int argc, char* argv[]) {
	//PROTOTIPI DI FUNZIONE
	__global__ void matrixMult (float *, float *, float *, int , int, int, int, int);
	void printMatrix(float *, const char[], int, int);

	//DICHIARAZIONE VARIABILI
	float *h_A=NULL, *h_B=NULL, *h_C=NULL, *d_A=NULL, *d_B=NULL, *d_C=NULL;
	int N=0, M=0, P=0, ntrow=0, ntcol=0, i=0;
	double inizio, fine;

	//CONTROLLI SUI PARAMETRI IN INGRESSO
	if(argc != 6){
		printf("Errore: eseguire nel seguente modo [N][M][P][ntrow][ntcol] ");
		return 0;
	}
	if(atoi(argv[1]) % atoi(argv[4]) != 0 || atoi(argv[3]) % atoi(argv[5]) != 0){
		printf("Errore: La matrice non puo' essere divisa in una griglia di %dx%d blocchi di %dx%d thread",(int)ceil(atoi(argv[1])/atoi(argv[4])),(int)ceil(atoi(argv[3])/atoi(argv[5])),atoi(argv[4]),atoi(argv[5]));
		return 0;
	}

	N = atoi(argv[1]);
	M = atoi(argv[2]);
	P = atoi(argv[3]);
	ntrow = atoi(argv[4]);
	ntcol = atoi(argv[5]);

	//INIZIALIZZAZIONE DELLE MATRICI
	h_A = (float *) malloc(N * M * sizeof(float));
	h_B = (float *) malloc(M * P * sizeof(float));
	h_C = (float *) malloc(N * P * sizeof(float));
	hipMalloc((void **) &d_A, N * M * sizeof(float));
	hipMalloc((void **) &d_B, M * P * sizeof(float));
	hipMalloc((void **) &d_C, N * P * sizeof(float));

	for(i=0; i < N * M; i++)
		*(h_A + i) = i+1;
	for(i=0; i < M * P; i++)
		*(h_B + i) = i+1;
	for(i=0; i < N * P; i++)
		*(h_C + i) = 0;

	//COPIA DELLE MATRICI SULLA GPU
	hipMemcpy(d_A, h_A, N * M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, M * P * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, N * P * sizeof(float), hipMemcpyHostToDevice);

	//DEFINZIONE DIMENSIONI DEI BLOCCHI
	dim3 dimBlock(ntcol, ntrow);
	dim3 dimGrid((int)ceil(P/dimBlock.x), (int)ceil(N/dimBlock.y));

	//ESECUZIONE DELL'ALGORITMO CON IL CALCOLO DEL TEMPO
	inizio = get_cur_time();
		matrixMult<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N, M, P, ntrow, ntcol);
    hipDeviceSynchronize();
	fine = get_cur_time();

	printf("GPU Computation Time: %lfs\n", fine - inizio);
	printf("Performance: %e\n",((double)2*N*M*P)/(fine-inizio));

	hipMemcpy(h_C, d_C, N * P * sizeof(float), hipMemcpyDeviceToHost);

	//STAMPA DELLE MATRICI
	//printMatrix(h_A, "A", N, M);
	//printf("\n");
	//printMatrix(h_B, "B", M, P);
	//printf("\n");
	//printMatrix(h_C, "C", N, P);
	//printf("\n");

	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}

__global__ void matrixMult(float* A, float* B, float* C,int N, int M, int P, int ntrow,int ntcol){
	int k, temp = 0;
	int col = blockIdx.x*ntcol + threadIdx.x;
	int row = blockIdx.y*ntrow + threadIdx.y;

	if(col < P && row < N) {
		for (k = 0; k < M; k++)
			temp += A[row * M + k] * B[k * P + col];
		C[row * P + col] = temp;
		//printf("%d|%d %d|%d C(%d,%d):%.2f\n",blockIdx.y,blockIdx.x,threadIdx.y,threadIdx.x, row, col, C[row*P+col]);
	}
} 

void printMatrix(float *M, const char name[], int row, int col) {
	int i;
	printf("%s:", name);
	for(i=0; i < row * col; i++) {
		if(i % col == 0) printf("\n");
		printf("%.2f ", *(M + i));
	}
	printf("\n");
}